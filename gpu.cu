#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <stdio.h>
#include <assert.h>
#include <math.h>
#include <hip/hip_runtime.h>
#include "common.h"

#define NUM_THREADS 256

int number, totalN;
double da;

extern double size;
//
//  benchmarking program
//



__global__ int place(double x, double y, double da, int number) {
    int xID = x / da;
    int yID = y / da;
    return xID * number + yID;
}

__global__ int place(particle_t &particle, double da, int number) {
    int xID = particle.x / da;
    int yID = particle.y / da;
    return xID * number + yID;
}

__global__ void assign_particles(int n, particle_t * particles, int* d_next, int* d_grids, double da, int number) {
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    if(tid >= n) return;

    int k = locationToID(particles[tid], da, number);
    d_next[tid] = atomicExch(&d_grids[k], tid);
}

__global__ void grids(int totalN, int* d_grids) {
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    if(tid >= totalN) return;

    d_grids[tid] = -1;
}



__device__ void apply_force_gpu(particle_t &particle, particle_t &neighbor)
{
    double dx = neighbor.x - particle.x;
    double dy = neighbor.y - particle.y;
    double r2 = dx * dx + dy * dy;
    if( r2 > cutoff*cutoff )
        return;
    //r2 = fmax( r2, min_r*min_r );
    r2 = (r2 > min_r*min_r) ? r2 : min_r*min_r;
    double r = sqrt( r2 );

    //
    //  very simple short-range repulsive force
    //
    double coef = ( 1 - cutoff / r ) / r2 / mass;
    particle.ax += coef * dx;
    particle.ay += coef * dy;

}

__global__ void compute_forces_gpu(particle_t * particles, int n)
{
    // Get thread (particle) ID
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    if(tid >= n) return;

    particles[tid].ax = particles[tid].ay = 0;
    for(int j = 0 ; j < n ; j++)
        apply_force_gpu(particles[tid], particles[j]);

}


__global__ void compute_grid_forces_gpu(particle_t * particles, int * d_next,int tot_num, int * d_grids, double dim, int num)
{
    // Get thread (particle) ID
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    if(tid >= tot_num) return;

    int xID = tid / num;
    int yID = tid % num;
    int k = tid;

    for(int i = d_grids[tid]; i != -1; i = d_next[i]) {
        particle_t * p = &particles[i];

        p->ax = p->ay = 0;

        // check self
        compute_self_grid_forces(i, particles, d_next, d_grids[k]);

        // check other
        if(xID > 0) {
            compute_grid_forces(i, particles, d_next, d_grids[k - num]);
            if(yID > 0)
                compute_grid_forces(i, particles, d_next, d_grids[k - num - 1]);
            if(yID < num - 1)
                compute_grid_forces(i, particles, d_next, d_grids[k - num + 1]);
        }
        if(xID < num - 1) {
            compute_grid_forces(i, particles, d_next, d_grids[k + num]);
            if(yID > 0)
                compute_grid_forces(i, particles, d_next, d_grids[k + num - 1]);
            if(yID < num - 1)
                compute_grid_forces(i, particles, d_next, d_grids[k + num + 1]);
        }
        if(yID > 0) compute_grid_forces(i, particles, d_next, d_grids[k - 1]);
        if(yID < num - 1) compute_grid_forces(i, particles, d_next, d_grids[k + 1]);
    }
}


__global__ void move_gpu (particle_t * particles, int n, double size)
{

    // Get thread (particle) ID
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    if(tid >= n) return;

    particle_t * p = &particles[tid];
    //
    //  slightly simplified Velocity Verlet integration
    //  conserves energy better than explicit Euler method
    //
    p->vx += p->ax * dt;
    p->vy += p->ay * dt;
    p->x  += p->vx * dt;
    p->y  += p->vy * dt;

    //
    //  bounce from walls
    //
    while( p->x < 0 || p->x > size )
    {
        p->x  = p->x < 0 ? -(p->x) : 2*size-p->x;
        p->vx = -(p->vx);
    }
    while( p->y < 0 || p->y > size )
    {
        p->y  = p->y < 0 ? -(p->y) : 2*size-p->y;
        p->vy = -(p->vy);
    }

}



int main( int argc, char **argv )
{
    // This takes a few seconds to initialize the runtime
    hipDeviceSynchronize();

    if( find_option( argc, argv, "-h" ) >= 0 )
    {
        printf( "Options:\n" );
        printf( "-h to see this help\n" );
        printf( "-n <int> to set the number of particles\n" );
        printf( "-o <filename> to specify the output file name\n" );
        return 0;
    }

    int n = read_int( argc, argv, "-n", 1000 );

    char *savename = read_string( argc, argv, "-o", NULL );

    FILE *fsave = savename ? fopen( savename, "w" ) : NULL;
    particle_t *particles = (particle_t*) malloc( n * sizeof(particle_t) );

    // GPU particle data structure
    particle_t * d_particles;
    hipMalloc((void **) &d_particles, n * sizeof(particle_t));

    set_size( n );

    init_particles( n, particles );

    hipDeviceSynchronize();
    double copy_time = read_timer( );

    // Copy the particles to the GPU
    hipMemcpy(d_particles, particles, n * sizeof(particle_t), hipMemcpyHostToDevice);

  //  hipDeviceSynchronize();
    copy_time = read_timer( ) - copy_time;

    //
    //  simulate a number of time steps
    //
   // hipDeviceSynchronize();      //commented out from original
    double simulation_time = read_timer( );

    for( int step = 0; step < NSTEPS; step++ )
    {
        //
        //  compute forces
        //

        int blks = (n + NUM_THREADS - 1) / NUM_THREADS;
        compute_forces_gpu <<< blks, NUM_THREADS >>> (d_particles, n);

        //
        //  move particles
        //
        move_gpu <<< blks, NUM_THREADS >>> (d_particles, n, size);

        //
        //  save if necessary
        //
        if( fsave && (step%SAVEFREQ) == 0 ) {
            // Copy the particles back to the CPU
            hipMemcpy(particles, d_particles, n * sizeof(particle_t), hipMemcpyDeviceToHost);
            save( fsave, n, particles);
        }
    }
   // hipDeviceSynchronize();        //may need to commet out to upspeed
    simulation_time = read_timer( ) - simulation_time;

    printf( "CPU-GPU copy time = %g seconds\n", copy_time);
    printf( "n = %d, simulation time = %g seconds\n", n, simulation_time );

    free( particles );
    hipFree(d_particles);
    if( fsave )
        fclose( fsave );

    return 0;
}